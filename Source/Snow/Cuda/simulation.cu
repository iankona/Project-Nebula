#include "hip/hip_runtime.h"
/**************************************************************************
**
**   SNOW - CS224 BROWN UNIVERSITY
**
**   simulation.cu
**   Authors: evjang, mliberma, taparson, wyegelwe
**   Created: 17 Apr 2014
**
**************************************************************************/

#define CUDA_INCLUDE

#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include "math.h"

#include "sim/caches.h"
#include "sim/implicitcollider.h"
#include "sim/material.h"
#include "sim/particle.h"
#include "sim/particlegridnode.h"

#include "common/math.h"

#include "cuda/helpers.h"
#include "cuda/atomic.h"
#include "cuda/collider.h"
#include "cuda/decomposition.h"
#include "cuda/implicit.h"
#include "cuda/weighting.h"

#include "cuda/functions.h"

#define ALPHA 0.05f

#define GRAVITY vec3(0.f,-9.8f,0.f)

// Chain to compute the volume of the particle
/**
 * Part of one time operation to compute particle volumes. First rasterize particle masses to grid
 *
 * Operation done over Particles over grid node particle affects
 */
__global__ void computeNodeMasses( const Particle *particles, int numParticles, const Grid *grid, float *nodeMasses )
{
    int particleIdx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
    if ( particleIdx >= numParticles ) return;

    const Particle &particle = particles[particleIdx];

    glm::ivec3 currIJK;
    Grid::gridIndexToIJK( threadIdx.y, glm::ivec3(4,4,4), currIJK );
    vec3 particleGridPos = (particle.position - grid->pos) / grid->h;
    currIJK += glm::ivec3(particleGridPos-1);

    if ( Grid::withinBoundsInclusive(currIJK, glm::ivec3(0,0,0), grid->dim) ) {
        vec3 nodePosition(currIJK);
        vec3 dx = vec3::abs( particleGridPos - nodePosition );
        float w = weight( dx );
        atomicAdd( &nodeMasses[Grid::getGridIndex(currIJK, grid->dim+1)], particle.mass*w );
     }
}

/**
 * Computes the particle's density * grid's volume. This needs to be separate from computeCellMasses(...) because
 * we need to wait for ALL threads to sync before computing the density
 *
 * Operation done over Particles over grid node particle affects
 */
__global__ void computeParticleDensity( Particle *particles, int numParticles, const Grid *grid, const float *cellMasses )
{
    int particleIdx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
    if ( particleIdx >= numParticles ) return;

    Particle &particle = particles[particleIdx];

    glm::ivec3 currIJK;
    Grid::gridIndexToIJK( threadIdx.y, glm::ivec3(4,4,4), currIJK );
    vec3 particleGridPos = ( particle.position - grid->pos ) / grid->h;
    currIJK += glm::ivec3(particleGridPos-1);

    if ( Grid::withinBoundsInclusive(currIJK, glm::ivec3(0,0,0), grid->dim) ) {
        vec3 nodePosition(currIJK);
        vec3 dx = vec3::abs( particleGridPos - nodePosition );
        float w = weight( dx );
        float gridVolume = grid->h * grid->h * grid->h;
        atomicAdd( &particle.volume, cellMasses[Grid::getGridIndex(currIJK, grid->dim+1)] * w / gridVolume ); //fill volume with particle density. Then in final step, compute volume
     }
}


/**
 * Computes the particle's volume. Assumes computeParticleDensity(...) has just been called.
 *
 * Operation done over particles
 */
__global__ void computeParticleVolume( Particle *particleData, int numParticles )
{
    int particleIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( particleIdx >= numParticles ) return;
    Particle &particle = particleData[particleIdx];
    particle.volume = particle.mass / particle.volume; // Note: particle.volume is assumed to be the (particle's density ) before we compute it correctly
}

__host__ void initializeParticleVolumes( Particle *particles, int numParticles, const Grid *grid, int numNodes )
{
    float *devNodeMasses;
    checkCudaErrors( hipMalloc( (void**)&devNodeMasses, numNodes*sizeof(float) ) );
    hipMemset( devNodeMasses, 0, numNodes*sizeof(float) );

    const dim3 blocks( (numParticles+THREAD_COUNT-1)/THREAD_COUNT, 64 );
    static const dim3 threads( THREAD_COUNT / 64, 64 );

    LAUNCH( computeNodeMasses<<<blocks,threads>>>(particles,numParticles,grid,devNodeMasses) );

    LAUNCH( computeParticleDensity<<<blocks,threads>>>(particles,numParticles,grid,devNodeMasses) );

    LAUNCH( computeParticleVolume<<<(numParticles+THREAD_COUNT-1)/THREAD_COUNT,THREAD_COUNT>>>(particles,numParticles) );

    checkCudaErrors( hipFree(devNodeMasses) );
}

__global__ void computeSigma( const Particle *particles, ParticleCache *particleCache, int numParticles, const Grid *grid )
{
    int particleIdx = blockIdx.x*blockDim.x + threadIdx.x;
    if ( particleIdx >= numParticles ) return;

    const Particle &particle = particles[particleIdx];

    const mat3 &Fp = particle.plasticF; //for the sake of making the code look like the math
    const mat3 &Fe = particle.elasticF;

    float Jpp = mat3::determinant(Fp);
    float Jep = mat3::determinant(Fe);

    mat3 Re;
    computePD( Fe, Re );

    const Material material = particle.material;

    float muFp = material.mu*expf(material.xi*(1-Jpp));
    float lambdaFp = material.lambda*expf(material.xi*(1-Jpp));

    particleCache->sigmas[particleIdx] = (2*muFp*mat3::multiplyABt(Fe-Re, Fe) + mat3(lambdaFp*(Jep-1)*Jep)) * -particle.volume;
}

/**
 * Called on each particle.
 *
 * Each particle adds it's mass, velocity and force contribution to the grid nodes within 2h of itself.
 *
 * In:
 * particleData -- list of particles
 * grid -- Stores grid paramters
 * worldParams -- Global parameters dealing with the physics of the world
 *
 * Out:
 * nodes -- list of every node in grid ((dim.x+1)*(dim.y+1)*(dim.z+1))
 *
 */
__global__ void computeCellMassVelocityAndForceFast( const Particle *particleData, const ParticleCache *particleCache, int numParticles, const Grid *grid, Node *nodes )
{
    int particleIdx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
    if ( particleIdx >= numParticles ) return;

    const Particle &particle = particleData[particleIdx];

    glm::ivec3 currIJK;
    Grid::gridIndexToIJK(threadIdx.y, glm::ivec3(4,4,4), currIJK);
    vec3 particleGridPos = (particle.position-grid->pos)/grid->h;
    currIJK += glm::ivec3( particleGridPos-1 );

    if ( Grid::withinBoundsInclusive(currIJK, glm::ivec3(0,0,0), grid->dim) ) {
        Node &node = nodes[Grid::getGridIndex(currIJK, grid->dim+1)];

        float w;
        vec3 wg;
        vec3 nodePosition(currIJK.x, currIJK.y, currIJK.z);
        weightAndGradient( particleGridPos - nodePosition, w, wg );

        atomicAdd( &node.mass, particle.mass*w );
        atomicAdd( &node.velocity, particle.velocity*particle.mass*w );
        atomicAdd( &node.force, particleCache->sigmas[particleIdx]*wg );
     }
}

/**
 * Called on each grid node.
 *
 * Updates the velocities of each grid node based on forces and collisions
 *
 * In:
 * nodes -- list of all nodes in the grid.
 * dt -- delta time, time step of simulation
 * colliders -- array of colliders in the scene.
 * numColliders -- number of colliders in the scene
 * worldParams -- Global parameters dealing with the physics of the world
 * grid -- parameters defining the grid
 *
 * Out:
 * nodes -- updated velocity and velocityChange
 *
 */
__global__ void updateNodeVelocities( Node *nodes, int numNodes, float dt, const ImplicitCollider* colliders, int numColliders, const Grid *grid, bool updateVelocityChange )
{
    int nodeIdx = blockIdx.x*blockDim.x + threadIdx.x;
    if ( nodeIdx >= numNodes ) return;

    Node &node = nodes[nodeIdx];

    if ( node.mass > 0.f ) {

        // Have to normalize velocity by mass to conserve momentum
        float scale = 1.f / node.mass;
        node.velocity *= scale;

        // Initialize velocityChange with pre-update velocity
        node.velocityChange = node.velocity;

        // Gravity for node forces
        node.force += node.mass * GRAVITY;

        // Update velocity with node force
        node.velocity += dt * scale * node.force;

        // Handle collisions
        int gridI, gridJ, gridK;
        Grid::gridIndexToIJK( nodeIdx, gridI, gridJ, gridK, grid->dim+1 );
        vec3 nodePosition = vec3(gridI, gridJ, gridK)*grid->h + grid->pos;
        checkForAndHandleCollisions( colliders, numColliders, nodePosition, node.velocity );

        if ( updateVelocityChange ) node.velocityChange = node.velocity - node.velocityChange;

    }
}

// Use weighting functions to compute particle velocity gradient and update particle velocity
__device__ void processGridVelocities( Particle &particle, const Grid *grid, const Node *nodes, mat3 &velocityGradient )
{
    const vec3 &pos = particle.position;
    const glm::ivec3 &dim = grid->dim;
    const float h = grid->h;

    // Compute neighborhood of particle in grid
    vec3 particleGridPos = (pos - grid->pos) / h,
         gridMax = vec3::floor( particleGridPos + vec3(2,2,2) ),
         gridMin = vec3::ceil( particleGridPos - vec3(2,2,2) );
    glm::ivec3 maxIndex = glm::clamp( glm::ivec3(gridMax), glm::ivec3(0,0,0), dim ),
               minIndex = glm::clamp( glm::ivec3(gridMin), glm::ivec3(0,0,0), dim );

    // For computing particle velocity gradient:
    //      grad(v_p) = sum( v_i * transpose(grad(w_ip)) ) = [3x3 matrix]
    // For updating particle velocity:
    //      v_PIC = sum( v_i * w_ip )
    //      v_FLIP = v_p + sum( dv_i * w_ip )
    //      v = (1-alpha)*v_PIC _ alpha*v_FLIP
    vec3 v_PIC(0,0,0), dv_FLIP(0,0,0);
    int rowSize = dim.z+1;
    int pageSize = (dim.y+1)*rowSize;
    for ( int i = minIndex.x; i <= maxIndex.x; ++i ) {
        vec3 d, s;
        d.x = particleGridPos.x - i;
        d.x *= ( s.x = ( d.x < 0 ) ? -1.f : 1.f );
        int pageOffset = i*pageSize;
        for ( int j = minIndex.y; j <= maxIndex.y; ++j ) {
            d.y = particleGridPos.y - j;
            d.y *= ( s.y = ( d.y < 0 ) ? -1.f : 1.f );
            int rowOffset = pageOffset + j*rowSize;
            for ( int k = minIndex.z; k <= maxIndex.z; ++k ) {
                d.z = particleGridPos.z - k;
                d.z *= ( s.z = ( d.z < 0 ) ? -1.f : 1.f );
                const Node &node = nodes[rowOffset+k];
                float w;
                vec3 wg;
                weightAndGradient( s, d, w, wg );
                velocityGradient += mat3::outerProduct( node.velocity, wg );
                // Particle velocities
                v_PIC += node.velocity * w;
                dv_FLIP += node.velocityChange * w;
            }
        }
    }
    particle.velocity = (1.f-ALPHA)*v_PIC + ALPHA*(particle.velocity+dv_FLIP);
}

__device__ void updateParticleDeformationGradients( Particle &particle, const mat3 &velocityGradient, float timeStep )
{
    // Temporarily assign all deformation to elastic portion
    particle.elasticF = mat3::addIdentity( timeStep*velocityGradient ) * particle.elasticF;
    const Material &material = particle.material;
    // Clamp the singular values
    mat3 W, S, Sinv, V;
    computeSVD( particle.elasticF, W, S, V );

    // FAST COMPUTATION:
    S = mat3( CLAMP( S[0], material.criticalCompressionRatio, material.criticalStretchRatio ), 0.f, 0.f,
              0.f, CLAMP( S[4], material.criticalCompressionRatio, material.criticalStretchRatio ), 0.f,
              0.f, 0.f, CLAMP( S[8], material.criticalCompressionRatio, material.criticalStretchRatio ) );
    Sinv = mat3( 1.f/S[0], 0.f, 0.f,
                 0.f, 1.f/S[4], 0.f,
                 0.f, 0.f, 1.f/S[8] );
    particle.plasticF = mat3::multiplyADBt( V, Sinv, W ) * particle.elasticF * particle.plasticF;
    particle.elasticF = mat3::multiplyADBt( W, S, V );

//     // MORE ACCURATE COMPUTATION:
//    S[0] = CLAMP( S[0], material->criticalCompressionRatio, material->criticalStretchRatio );
//    S[4] = CLAMP( S[4], material->criticalCompressionRatio, material->criticalStretchRatio );
//    S[8] = CLAMP( S[8], material->criticalCompressionRatio, material->criticalStretchRatio );
//    particle.plasticF = V * mat3::inverse( S ) * mat3::transpose( W ) * particle.elasticF * particle.plasticF;
//    particle.elasticF = W * S * mat3::transpose( V );

}

__global__ void updateParticlesFromGrid( Particle *particles, int numParticles, const Grid *grid, const Node *nodes, float timeStep, const ImplicitCollider *colliders, int numColliders )
{
    int particleIdx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( particleIdx >= numParticles ) return;

    Particle &particle = particles[particleIdx];

    // Update particle velocities and fill in velocity gradient for deformation gradient computation
    mat3 velocityGradient = mat3( 0.f );
    processGridVelocities( particle, grid, nodes, velocityGradient );

    updateParticleDeformationGradients( particle, velocityGradient, timeStep );

    checkForAndHandleCollisions( colliders, numColliders, particle.position, particle.velocity );

    particle.position += timeStep * ( particle.velocity );
}

__global__ void updateColliderPositions(ImplicitCollider *colliders, int numColliders,float timestep)
{
    int colliderIdx = blockDim.x*blockIdx.x + threadIdx.x;
    colliders[colliderIdx].center += colliders[colliderIdx].velocity*timestep;
}

__host__ void updateParticles( Particle *particles, ParticleCache *devParticleCache, ParticleCache *hostParticleCache, int numParticles,
                               Grid *grid, Node *nodes, NodeCache *nodeCaches, int numNodes,
                               ImplicitCollider *colliders, int numColliders,
                               float timeStep, bool implicitUpdate )
{
    hipDeviceSetCacheConfig( hipFuncCachePreferL1 );

    // Clear data before update
    checkCudaErrors( hipMemset(nodes, 0, numNodes*sizeof(Node)) );
    checkCudaErrors( hipMemset(nodeCaches, 0, numNodes*sizeof(NodeCache)) );

    // All dat ParticleCache data
    hipMemset( hostParticleCache->sigmas, 0, numParticles*sizeof(mat3) );
    hipMemset( hostParticleCache->Aps, 0, numParticles*sizeof(mat3) );
    hipMemset( hostParticleCache->FeHats, 0, numParticles*sizeof(mat3) );
    hipMemset( hostParticleCache->ReHats, 0, numParticles*sizeof(mat3) );
    hipMemset( hostParticleCache->SeHats, 0, numParticles*sizeof(mat3) );
    hipMemset( hostParticleCache->dFs, 0, numParticles*sizeof(mat3) );

    const dim3 pBlocks1D( (numParticles+THREAD_COUNT-1)/THREAD_COUNT );
    const dim3 nBlocks1D( (numNodes+THREAD_COUNT-1)/THREAD_COUNT );
    const dim3 threads1D( THREAD_COUNT );
    const dim3 pBlocks2D( (numParticles+THREAD_COUNT-1)/THREAD_COUNT, 64 );
    const dim3 threads2D( THREAD_COUNT/64, 64 );

    LAUNCH( updateColliderPositions<<<numColliders,1>>>(colliders,numColliders,timeStep) );

    LAUNCH( computeSigma<<<pBlocks1D,threads1D>>>(particles,devParticleCache,numParticles,grid) );

    LAUNCH( computeCellMassVelocityAndForceFast<<<pBlocks2D,threads2D>>>(particles,devParticleCache,numParticles,grid,nodes) );

    LAUNCH( updateNodeVelocities<<<nBlocks1D,threads1D>>>(nodes,numNodes,timeStep,colliders,numColliders,grid,!implicitUpdate) );

    if ( implicitUpdate ) integrateNodeForces( particles, devParticleCache, numParticles, grid, nodes, nodeCaches, numNodes, timeStep );

    LAUNCH( updateParticlesFromGrid<<<pBlocks1D,threads1D>>>(particles,numParticles,grid,nodes,timeStep,colliders,numColliders) );
}
