#include "hip/hip_runtime.h"
#include <stdio.h>

#include "Noise.h"

__global__ void Calc(int a, int b, int *c)
{
	//for (int i = 0; i < 9999999999; ++i)
	{
		*c = a + b;
	}
}

int test()
{
	int a, b, c;
	int *dev_c;
	scanf("%d%d", &a, &b);
	hipMalloc(&dev_c, sizeof(int));
	Calc<<<1, 1>>>(a, b, dev_c);
	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	printf("%d\n", c);
	printf("%f\n", CLAMP(0.1f, 0.0f, 1.0f));
	system("pause");
	return 0;

}