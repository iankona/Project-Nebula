#include <windows.h>
#include <cuda_gl_interop.h>
#include <Snow/SnowParticle.h>
#include "Functions.h"

void registerVBO( hipGraphicsResource **resource, GLuint vbo )
{
    checkCudaErrors( hipGraphicsGLRegisterBuffer(resource, vbo, cudaGraphicsMapFlagsNone) );
}

void unregisterVBO( hipGraphicsResource *resource )
{
    checkCudaErrors( hipGraphicsUnregisterResource(resource) );
}